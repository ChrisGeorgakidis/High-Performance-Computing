/*
* This sample implements a separable convolution
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

unsigned int filter_radius;

#define FILTER_LENGTH (2 * filter_radius + 1)
#define ABS(val) ((val) < 0.0 ? (-(val)) : (val))
#define accuracy 0.00005
#define ArraySize imageW *imageH
#define ERROR -1
#define FILTER_R_X2 2*filter_radius
#define SH_MEM_SIZE 32
#define NUMBLOCKS 4

typedef float dataType;
__constant__ dataType d_Filter[65536/sizeof(dataType)];

// This checks for cuda errors
#define cudaErrorCheck()                                                                                 \
  {                                                                                                      \
    hipError_t error = hipGetLastError();                                                              \
    if (error != hipSuccess)                                                                            \
    {                                                                                                    \
      printf("Cuda Error Found %s:%d:  '%s'\n", __FILE__, __LINE__, hipGetErrorString(error));          \
      freeMemory(h_Filter, h_Input, h_Buffer, h_OutputCPU, h_OutputGPU, d_Input, d_Buffer, d_OutputGPU); \
      return (ERROR);                                                                                    \
    }                                                                                                    \
  }

#define cudaCalloc(pointer, size, sizeOfElement)                                                         \
  {                                                                                                      \
    hipError_t err = hipMalloc(pointer, size * sizeOfElement);                                         \
    if (err != hipSuccess)                                                                              \
    {                                                                                                    \
      printf("Error allocating memory on host:   %s\n", hipGetErrorString(err));                        \
      freeMemory(h_Filter, h_Input, h_Buffer, h_OutputCPU, h_OutputGPU, d_Input, d_Buffer, d_OutputGPU); \
      return (ERROR);                                                                                    \
    }                                                                                                    \
    hipMemset(*pointer, 0.0, size *sizeOfElement);                                                      \
  }

////////////////////////////////////////////////////////////////////////////////
// Kernel Row Convolution Filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionRow(dataType *Input, dataType *Output, int filterR, int imageW)
{
  dataType sum = 0;
  int d, k;

  int ix = blockIdx.x * blockDim.x + threadIdx.x + filterR;
  int iy = blockIdx.y * blockDim.y + threadIdx.y + filterR;
  //int dimx = blockDim.x * gridDim.x + 2 * filterR;
  //int idx = iy * dimx + ix;
  int imageWithPaddingW = imageW + 2 * filterR;

  for (k = -filterR; k <= filterR; k++)
  {
    d = ix + k;
    sum += Input[iy * imageWithPaddingW + d] * d_Filter[filterR - k];
  }
  Output[iy * imageWithPaddingW + ix] = sum; //Only 1 time for each thread
}


////////////////////////////////////////////////////////////////////////////////
// Kernel Column Convolution Filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionColumn(dataType *Input, dataType *Output, int filterR, int imageW, int imageH)
{
  dataType sum = 0;
  int d, k;

  int ix = blockIdx.x * blockDim.x + threadIdx.x + filterR;
  int iy = blockIdx.y * blockDim.y + threadIdx.y + filterR;

  int imageWithPaddingW = imageW + 2 * filterR;

  for (k = -filterR; k <= filterR; k++)
  {
    d = iy + k;

    sum += Input[d * imageWithPaddingW + ix] * d_Filter[filterR - k];

    Output[iy * imageWithPaddingW + ix] = sum;
  }
}


////////////////////////////////////////////////////////////////////////////////
// Kernel Row Convolution Filter using Shared Memory
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionRowSharedMem(dataType *Input, dataType *Output, int filterR, int imageW, int SH_MEM_SIZE_PAD)
{
  dataType sum = 0;
  int d, k;

  int tx = threadIdx.x + filterR;
  int ix = blockIdx.x * blockDim.x + threadIdx.x + filterR;
  int iy = blockIdx.y * blockDim.y + threadIdx.y + filterR;

  //indexes in arrays including padding
  int ptx = threadIdx.x;
  int pty = threadIdx.y;
  int pix = blockIdx.x * blockDim.x + threadIdx.x;

  int imageWithPaddingW = imageW + 2 * filterR;

  //shared memory for Input
  extern __shared__ dataType s_Input[]; // shared memory with padding

  //collaboratively load tiles into __shared__
  for (int i = 0; i < SH_MEM_SIZE_PAD/32; i++){
    s_Input[pty * SH_MEM_SIZE_PAD + (SH_MEM_SIZE_PAD / 32) * ptx + i] = Input[iy * imageWithPaddingW + (SH_MEM_SIZE_PAD / 32) * pix + i - ((SH_MEM_SIZE_PAD / 32) - 1) * (blockIdx.x * blockDim.x)];
  }

  __syncthreads();

  for (k = -filterR; k <= filterR; k++)
  {
    d = tx + k;
    sum += s_Input[pty * SH_MEM_SIZE_PAD + d] * d_Filter[filterR - k];
  }
  Output[iy * imageWithPaddingW + ix] = sum; //1 time for each thread
}


////////////////////////////////////////////////////////////////////////////////
// Kernel Column Convolution Filter using Shared Memory
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionColumnSharedMem(dataType *Input, dataType *Output, int filterR, int imageW, int imageH, int SH_MEM_SIZE_PAD)
{
  dataType sum = 0;
  int d, k;

  int ty = threadIdx.y + filterR;
  int ix = blockIdx.x * blockDim.x + threadIdx.x + filterR;
  int iy = blockIdx.y * blockDim.y + threadIdx.y + filterR;

  //indexes in arrays including padding
  int ptx = threadIdx.x;
  int pty = threadIdx.y;
  int piy = blockIdx.y * blockDim.y + pty;

  int imageWithPaddingW = imageW + 2 * filterR;

  //shared memory for Input
  extern __shared__ dataType s_Input[];

  //collaboratively load tiles into __shared__
  for (int i = 0; i < SH_MEM_SIZE_PAD/32; i++){
    s_Input[(pty * (SH_MEM_SIZE_PAD / 32) + i) * SH_MEM_SIZE + ptx] = Input[(piy * (SH_MEM_SIZE_PAD / 32) + i - ((SH_MEM_SIZE_PAD / 32) - 1)*(blockIdx.y * blockDim.y)) * imageWithPaddingW + ix];
  }
  __syncthreads();

  for (k = -filterR; k <= filterR; k++)
  {
    d = ty + k;
    sum += s_Input[d * SH_MEM_SIZE + ptx] * d_Filter[filterR - k];
  }
  Output[iy * imageWithPaddingW + ix] = sum; //One time for each thread
}


////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(dataType *h_Dst, dataType *h_Src, dataType *h_Filter,
int imageW, int imageH, int filterR)
{
  int x, y, k;
  int imageWithPaddingW = imageW + 2 * filterR;

  for (y = filterR; y < (imageH + filterR); y++)
  {
    for (x = filterR; x < (imageW + filterR); x++)
    {
      dataType sum = 0;

      for (k = -filterR; k <= filterR; k++)
      {
        int d = x + k;
        sum += h_Src[y * imageWithPaddingW + d] * h_Filter[filterR - k];
      }
      h_Dst[y * imageWithPaddingW + x] = sum; //One time for each x & y
    }
  }
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(dataType *h_Dst, dataType *h_Src, dataType *h_Filter,
int imageW, int imageH, int filterR)
{
  int x, y, k;
  int imageWithPaddingW = imageW + 2 * filterR;

  for (y = filterR; y < (imageH + filterR); y++)
  {
    for (x = filterR; x < (imageW + filterR); x++)
    {
      dataType sum = 0;

      for (k = -filterR; k <= filterR; k++)
      {
        int d = y + k;
        sum += h_Src[d * imageWithPaddingW + x] * h_Filter[filterR - k];
      }
      h_Dst[y * imageWithPaddingW + x] = sum; //One time for each x & y
    }
  }
}

////////////////////////////////////////////////////////////////////////////////
// Free Alocated Host and Device Memory
////////////////////////////////////////////////////////////////////////////////
int freeMemory(dataType *h_Filter, dataType *h_Input, dataType *h_Buffer, dataType *h_OutputCPU, dataType *h_OutputGPU, dataType *d_Input, dataType *d_Buffer, dataType *d_OutputGPU)
{
  hipError_t err;

  // free all the allocated memory for the host
  printf("Free host memory...\n");
  if (h_OutputGPU != NULL)
  {
    free(h_OutputGPU);
  }
  if (h_OutputCPU != NULL)
  {
    free(h_OutputCPU);
  }
  if (h_Buffer != NULL)
  {
    free(h_Buffer);
  }
  if (h_Input != NULL)
  {
    free(h_Input);
  }
  if (h_Filter != NULL)
  {
    free(h_Filter);
  }

  //free all the allocated device (GPU) memory
  printf("Free device memory...\n");
  if (d_OutputGPU != NULL)
  {
    err = hipFree(d_OutputGPU);
    if (err != hipSuccess)
    {
      printf("Error during hipFree (d_OutputGPU):  %s\n", hipGetErrorString(err));
      return (ERROR);
    }
  }
  if (d_Buffer != NULL)
  {
    err = hipFree(d_Buffer);
    if (err != hipSuccess)
    {
      printf("Error during hipFree (d_Buffer):  %s\n", hipGetErrorString(err));
      return (ERROR);
    }
  }
  if (d_Input != NULL)
  {
    err = hipFree(d_Input);
    if (err != hipSuccess)
    {
      printf("Error during hipFree (d_Input):  %s\n", hipGetErrorString(err));
      return (ERROR);
    }
  }

  // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
  printf("Reset Device\n");
  err = hipDeviceReset();
  if (err != hipSuccess)
  {
    printf("Error during cudaDeviceReset:  %s\n", hipGetErrorString(err));
    return (ERROR);
  }

  return (0);
}

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{

  //pointers for the host
  dataType
      *h_Filter = NULL,
      *h_Input = NULL,
      *h_Buffer = NULL,
      *h_OutputCPU = NULL,
      *h_OutputGPU = NULL;

  //pointers for the device
  dataType
      *d_Input = NULL,
      *d_Buffer = NULL,
      *d_OutputGPU = NULL;

  int imageW; //image width = N
  int imageH; //image height = N
  unsigned int i, j, block_size, numberOfBlocks;
  dataType diff = 0, max_diff = 0;

  /*-------timing variables-------*/
  struct timespec tv1, tv2;
  hipError_t err;
  hipEvent_t start;
  hipEvent_t stop;
  float elapsed;
  /*------------------------------*/

  /*------padding variables-------*/
  int imageWithPaddingW, newImageSize;
  /*------------------------------*/

  printf("Enter filter radius : ");
  scanf("%d", &filter_radius);

  // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
  // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
  // Gia aplothta thewroume tetragwnikes eikones.

  printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
  scanf("%d", &imageW); //TODO Warning
  imageH = imageW;

  imageWithPaddingW = imageW + 2 * filter_radius;
  newImageSize = imageWithPaddingW * imageWithPaddingW;

  printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
  printf("Image Width x Height = %i x %i\n\n", imageWithPaddingW, imageWithPaddingW);
  printf("Allocating and initializing host arrays...\n");

  //Allocate host (CPU) memory
  {
    h_Filter = (dataType *)malloc(FILTER_LENGTH * sizeof(dataType));
    if (h_Filter == NULL)
    {
      printf("Error allocating memory on host for h_Filter");
      freeMemory(h_Filter, h_Input, h_Buffer, h_OutputCPU, h_OutputGPU, d_Input, d_Buffer, d_OutputGPU);
      return (ERROR);
    }
    h_Input = (dataType *)calloc(newImageSize, sizeof(dataType));
    if (h_Input == NULL)
    {
      printf("Error allocating memory on host for h_Input");
      freeMemory(h_Filter, h_Input, h_Buffer, h_OutputCPU, h_OutputGPU, d_Input, d_Buffer, d_OutputGPU);
      return (ERROR);
    }
    h_Buffer = (dataType *)calloc(newImageSize, sizeof(dataType));
    if (h_Buffer == NULL)
    {
      printf("Error allocating memory on host for h_Buffer");
      freeMemory(h_Filter, h_Input, h_Buffer, h_OutputCPU, h_OutputGPU, d_Input, d_Buffer, d_OutputGPU);
      return (ERROR);
    }
    h_OutputCPU = (dataType *)calloc(newImageSize, sizeof(dataType));
    if (h_OutputCPU == NULL)
    {
      printf("Error allocating memory on host for h_OutputCPU");
      freeMemory(h_Filter, h_Input, h_Buffer, h_OutputCPU, h_OutputGPU, d_Input, d_Buffer, d_OutputGPU);
      return (ERROR);
    }
    h_OutputGPU = (dataType *)calloc(newImageSize, sizeof(dataType));
    if (h_OutputGPU == NULL)
    {
      printf("Error allocating memory on host for h_OutputGPU");
      freeMemory(h_Filter, h_Input, h_Buffer, h_OutputCPU, h_OutputGPU, d_Input, d_Buffer, d_OutputGPU);
      return (ERROR);
    }

    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.
  }

  printf("Allocate device (GPU) memory\n");
  //Allocate device (GPU) memory
  {
    err = hipMalloc((void **)&d_Input, newImageSize * sizeof(dataType));
    if (err != hipSuccess)
    {
      printf("Error allocating memory on host for d_Input:   %s\n", hipGetErrorString(err));
      freeMemory(h_Filter, h_Input, h_Buffer, h_OutputCPU, h_OutputGPU, d_Input, d_Buffer, d_OutputGPU);
      return (ERROR);
    }
    cudaCalloc((void **)&d_Buffer, newImageSize, sizeof(dataType));

    cudaCalloc((void **)&d_OutputGPU, newImageSize, sizeof(dataType));
  }

  if (imageW < 32)
  {
    block_size = imageW;
    numberOfBlocks = 1;
  }
  else
  {
    block_size = 32;
    numberOfBlocks = imageW / block_size;
  }

  dim3 threadsPerBlock(block_size, block_size);   //geometry for block
  dim3 numBlocks(numberOfBlocks, numberOfBlocks); //geometry for grid
  int SH_MEM_SIZE_PAD = 32 + 2 * filter_radius;

  //Initializations
  {
    srand(200);
    // Random initialization of h_Filter
    for (i = 0; i < FILTER_LENGTH; i++)
    {
      h_Filter[i] = (dataType)(rand() % 16);
    }

    // Random initialization of h_Input
    for (i = filter_radius; i < (imageH + filter_radius); i++)
    {
      for (j = filter_radius; j < (imageW + filter_radius); j++)
      {
        h_Input[i * imageWithPaddingW + j] = (dataType)rand() / ((dataType)RAND_MAX / 255) + (dataType)rand() / (dataType)RAND_MAX;
      }
    }
  }

  //CPU Computation
  {
    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation is about to start...\n");
    //Get the starting time
    clock_gettime(CLOCK_MONOTONIC_RAW, &tv1);
    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius);        // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles
    //Take the end time
    clock_gettime(CLOCK_MONOTONIC_RAW, &tv2);

    printf("CPU computation finished...\n");
  }

  //Calculate the duration of the CPU computation and report it
  {
    printf("\033[1;33m");
    printf("CPU time = %10g seconds\n",
           (double)(tv2.tv_nsec - tv1.tv_nsec) / 1000000000.0 +
               (double)(tv2.tv_sec - tv1.tv_sec));
  }
  printf("\033[0m");

  //Copy from host to device
  {
    hipEventCreate(&start);
    hipEventCreate(&stop);
    printf("Copy host memory to device\n");

    hipEventRecord(start, 0);
    //Copy host memory to device
    err = hipMemcpyToSymbol(HIP_SYMBOL(d_Filter), h_Filter, FILTER_LENGTH * sizeof(dataType));
    if (err != hipSuccess)
    {
      printf("Error during hipMemcpyToSymbol of h_Filter to d_Filter:  %s\n", hipGetErrorString(err));
      freeMemory(h_Filter, h_Input, h_Buffer, h_OutputCPU, h_OutputGPU, d_Input, d_Buffer, d_OutputGPU);
      return (ERROR);
    }
    err = hipMemcpy(d_Input, h_Input, newImageSize * sizeof(dataType), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
      printf("Error during hipMemcpy of h_Input to d_Input:  %s\n", hipGetErrorString(err));
      freeMemory(h_Filter, h_Input, h_Buffer, h_OutputCPU, h_OutputGPU, d_Input, d_Buffer, d_OutputGPU);
      return (ERROR);
    }
  }

  //GPU Computation
  {
    printf("GPU computation is about to start...\n");

    //kernel for row convolution
    //execute grid of numBlocks blocks of threadsPerBlock threads each
    convolutionRowSharedMem<<<numBlocks, threadsPerBlock, (32 *( 32 + 2 * filter_radius)) * sizeof(dataType)>>>(d_Input, d_Buffer, filter_radius, imageW, SH_MEM_SIZE_PAD);

    err = hipDeviceSynchronize();
    if (err != hipSuccess)
    {
      printf("Error during cudaDeviceSynchronize:  %s\n", hipGetErrorString(err));
      freeMemory(h_Filter, h_Input, h_Buffer, h_OutputCPU, h_OutputGPU, d_Input, d_Buffer, d_OutputGPU);
      return (ERROR);
    }

    //Error Checking
    cudaErrorCheck();

    //kernel for column convolution
    //execute grid of numBlocks blocks of threadsPerBlock threads each
    convolutionColumnSharedMem<<<numBlocks, threadsPerBlock, (32 * (32 + 2 * filter_radius)) * sizeof(dataType)>>>(d_Buffer, d_OutputGPU, filter_radius, imageW, imageH, SH_MEM_SIZE_PAD);

    err = hipMemcpy(h_OutputGPU, d_OutputGPU, newImageSize * sizeof(dataType), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
      printf("Error during hipMemcpy of d_OutputGPU to h_OutputGPU:  %s\n", hipGetErrorString(err));
      freeMemory(h_Filter, h_Input, h_Buffer, h_OutputCPU, h_OutputGPU, d_Input, d_Buffer, d_OutputGPU);
      return (ERROR);
    }

    //Error Checking
    cudaErrorCheck();
    hipEventRecord(stop, 0);
    printf("GPU computation finished...\n");
  }

  //Execution Time of GPU
  {
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    printf("\033[1;35m");
    printf("GPU time = %g ms\n", elapsed);
    printf("\033[0m");
    hipEventDestroy(start);
    hipEventDestroy(stop);
  }

  //Compare the results from CPU and GPU
  {
    for (i = filter_radius; i < imageH + filter_radius; i++)
    {
      for (j = filter_radius; j < imageW + filter_radius; j++)
      {
        diff = ABS(h_OutputCPU[i * imageWithPaddingW + j] - h_OutputGPU[i * imageWithPaddingW + j]);
        //printf("The difference between h_OutputCPU[%d]=%lf and h_OutputGPU[%d]=%lf is diff = %g\n", i * imageWithPaddingW + j, h_OutputCPU[i * imageWithPaddingW + j], i * imageWithPaddingW + j, h_OutputGPU[i * imageWithPaddingW + j], diff);
        if (diff > max_diff)
        {
          max_diff = diff;
        }
        if (diff > accuracy)
        {
          //printf("\t|->ERROR: The difference between the values of h_OutputCPU and h_OutputGPU at index i = %u is bigger than the given accuracy.\n", i);
        }
      }
    }

    if (max_diff > accuracy)
    {
      printf("\033[1;31m");
      printf("ERROR! Max difference between the values of h_OutputCPU and h_OutputGPU is max_diff = %g\n", max_diff);
    }
    else
    {
      printf("\033[1;32m");
      printf("Max difference between the values of h_OutputCPU and h_OutputGPU is max_diff = %g\n", max_diff);
    }
  }

  //Free allocated host and device memory
  printf("\033[0m");
  freeMemory(h_Filter, h_Input, h_Buffer, h_OutputCPU, h_OutputGPU, d_Input, d_Buffer, d_OutputGPU);

  return 0;
}
